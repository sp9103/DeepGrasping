#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#include <opencv2\opencv.hpp>

namespace caffe {

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;

  /////////////////////////////////////////�� �հ��� �� euclidean distance////////////////////////////////////////
  static int learningCount = 0;
  static Dtype UpperLeft = 0;
  static Dtype UpperRight = 0;
  static Dtype Thumb = 0;
  const int printStep = 10;
  int batchsize = bottom[0]->shape()[0];
  Dtype labelarr[1600], outputarr[1600], diffarr[1600];

 /* for (int i = 0; i < batchsize; i++){
	  Dtype endeffectorBox[9];
	  hipMemcpy(endeffectorBox, &diff_.gpu_data()[9 * i], sizeof(Dtype) * 9, hipMemcpyDeviceToHost);
	  UpperLeft += sqrt(endeffectorBox[0] * endeffectorBox[0] + endeffectorBox[1] * endeffectorBox[1] + endeffectorBox[2] * endeffectorBox[2]) / (Dtype)batchsize;
	  UpperRight += sqrt(endeffectorBox[3] * endeffectorBox[3] + endeffectorBox[4] * endeffectorBox[4] + endeffectorBox[5] * endeffectorBox[5]) / (Dtype)batchsize;
	  Thumb += sqrt(endeffectorBox[6] * endeffectorBox[6] + endeffectorBox[7] * endeffectorBox[7] + endeffectorBox[8] * endeffectorBox[8]) / (Dtype)batchsize;
  }

  if (learningCount == printStep-1){
	  printf("UpperLeft : %f, UpperRight : %f, Thumb : %f\n", UpperLeft / (Dtype)printStep, UpperRight / (Dtype)printStep, Thumb / (Dtype)printStep);
	  UpperLeft = UpperRight = Thumb = 0;
	  learningCount = 0;
  }

  learningCount = (learningCount+1) % 200;*/

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  if (std::isnan(loss) || std::isinf(loss)){
	  printf("loss invalide Error!\n");

	   for (int c = 0; c < bottom[0]->num(); c++){
	    hipMemcpy(labelarr, &bottom[1]->gpu_data()[9 * c], sizeof(Dtype) * 9, hipMemcpyDeviceToHost);
	    hipMemcpy(outputarr, &bottom[0]->gpu_data()[9 * c], sizeof(Dtype) * 9, hipMemcpyDeviceToHost);
	    hipMemcpy(diffarr, &diff_.gpu_data()[9 * c], sizeof(Dtype) * 9, hipMemcpyDeviceToHost);
	   }
  }

  //if (!std::isnan(loss)){

	 // //////////////////////////////////////////
	 // cv::Mat lossLayer;
	 // const int labelwidth = 60;
	 // const int labelheight = 60;
	 // lossLayer.create(labelheight, labelwidth * 3, CV_32FC1);

	 // ////////////label & output ���
	 // int cCount = bottom[0]->num() < 10 ? bottom[0]->num() : 10;
	 // for (int c = 0; c < cCount; c++){
		//  char buf[32];
		//  Dtype labelarr[3600], outputarr[3600], diffarr[3600];
		//  hipMemcpy(labelarr, &bottom[1]->gpu_data()[3600 * c], sizeof(Dtype) * 3600, hipMemcpyDeviceToHost);
		//  hipMemcpy(outputarr, &bottom[0]->gpu_data()[3600 * c], sizeof(Dtype) * 3600, hipMemcpyDeviceToHost);
		//  hipMemcpy(diffarr, &diff_.gpu_data()[3600 * c], sizeof(Dtype) * 3600, hipMemcpyDeviceToHost);

		//  for (int j = 0; j < labelheight * labelwidth; j++){
		//	  lossLayer.at<float>(j / labelwidth, j % labelwidth) = (float)labelarr[j];
		//	  lossLayer.at<float>(j / labelwidth, j % labelwidth + labelwidth) = (float)outputarr[j];
		//  }

		//  Dtype max = -9999;
		//  Dtype min = 9999;
		//  
		//  for (int j = 0; j < labelheight * labelwidth; j++){
		//	  if (max < diffarr[j])		max = diffarr[j];
		//	  if (min > diffarr[j])		min = diffarr[j];
		//  }

		//  for (int j = 0; j < labelheight * labelwidth; j++)
		//	  lossLayer.at<float>(j / labelwidth, j % labelwidth + labelwidth * 2) = (float)((diffarr[j] - min) / (max - min));

		//  sprintf(buf, "Loss_%d", c);
		//  cv::imshow(buf, lossLayer);
		//  cv::waitKey(0);
	 // }
  ////}
  //cv::destroyAllWindows();
  ////////////////////////////////////////////////////////
}

//Diff 0������ ���ְ� 1������ ����
template <typename Dtype>
void EuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
	  if (propagate_down[i]) {
		  const Dtype sign = (i == 0) ? 1 : -1;
		  const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
		  //const Dtype alpha = 1.0 / bottom[i]->num();
		  Dtype tt = top[0]->cpu_diff()[0];
		  caffe_gpu_axpby(
			  bottom[i]->count(),              // count
			  alpha,                              // alpha
			  diff_.gpu_data(),                   // a			//���̰��� diff_�� �����
			  Dtype(0),                           // beta
			  bottom[i]->mutable_gpu_diff());  // b
	  }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EuclideanLossLayer);

}  // namespace caffe
