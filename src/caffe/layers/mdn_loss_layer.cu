#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#include <opencv2\opencv.hpp>

#define MATH_PI		3.14159265

namespace caffe {

template <typename Dtype>							//mu_ik - tk calculation
__global__ void kernel_label_subtract(const int count,
	const int param_size, const int class_size, const int data_dim,
	const Dtype* data, const Dtype* label, Dtype* diff) {
	CUDA_KERNEL_LOOP(index, count) {
		int internal_idx = index % data_dim;					//mu vector���� ���° �ε���
		int outer_idx = index / data_dim;						//���° Ŭ����
		int label_idx = index / (class_size * data_dim);		//���° label == ���° batch
		diff[index] = data[outer_idx * param_size + internal_idx + 1] - label[label_idx * data_dim + internal_idx];
	}
}

template <typename Dtype>							// || mu-t || ^ 2
__global__ void kernel_diff_norm(const int count,
	const int class_size, const int data_dim,
	const Dtype* diff_squre, Dtype* norm) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype sum = 0;
		for (int i = 0; i < data_dim; i++)
			sum += diff_squre[index * data_dim + i];
		norm[index] = sum;
	}
}

template <typename Dtype>							// alpha * gaussian distribution ���
__global__ void kernel_normal_distribution(const int count,
	const int param_size, const int class_size, const int data_dim,
	const Dtype* norm, const Dtype* data, Dtype* alpha_distribution) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype alpha = data[index*param_size];
		Dtype sigma = data[index*param_size + 1 + data_dim];
		Dtype exp_gaussian = exp(- norm[index] / sigma / sigma / 2);
		Dtype distribution = exp_gaussian / pow(sigma, data_dim) / pow(2 * MATH_PI, data_dim / 2);
		//alpha * gaussian_distribution;
		alpha_distribution[index] = alpha * distribution;
	}
}

template <typename Dtype>							// ��(alpha * gaussian distribution) ���
__global__ void kernel_class_summation(const int count, const int class_size,
	const Dtype* alpha_pi_, Dtype* alpha_pi_sum_) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype sum = 0;
		for (int i = 0; i < class_size; i++)
			sum += alpha_pi_[index * class_size + i];
		alpha_pi_sum_[index] = sum;
	}
}

template <typename Dtype>							// posterior calculation 
__global__ void kernel_posterior_calc(const int count,
	const int batch_size, const int class_size,
	const Dtype* alpha_pi_, const Dtype* alpha_pi_sum_, Dtype* posterior) {
	CUDA_KERNEL_LOOP(index, count) {
		const int batch_idx = index / class_size;
		posterior[index] = alpha_pi_[index] / alpha_pi_sum_[batch_idx];
	}
}

template <typename Dtype>							// backpropagation delta calculation 
__global__ void kernel_delta_calc(const int count,
	const int batch_size, const int class_size, const int param_size, const int data_dim, 
 	const Dtype* posterior, const Dtype* diff, const Dtype* diff_norm, const Dtype* bottom_data, Dtype* bottom_diff) {
	CUDA_KERNEL_LOOP(index, count) {
 		const int internal_idx = index % param_size;
		const int class_idx = index / param_size;
		const Dtype sigma = bottom_data[class_idx*param_size + param_size - 1];
		if (internal_idx == 0){							//alpha delta calculate
			bottom_diff[index] = bottom_data[index] - posterior[class_idx];
		}
		else if (internal_idx == param_size - 1){		//sigma delta calculate
			bottom_diff[index] = -posterior[class_idx] * (diff_norm[class_idx] / sigma / sigma - data_dim);
		}
		else{											//mu delta calculate
			const int data_idx = internal_idx - 1;		//[0, datadim-1]
			Dtype diff_ik = diff[data_dim * class_idx + data_idx];
			bottom_diff[index] = posterior[class_idx] * (diff_ik / sigma / sigma);
		}
	}
}

template <typename Dtype>
void MDNLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

	const Dtype* bottom_data = bottom[0]->gpu_data();
	const Dtype* label = bottom[1]->gpu_data();
	const int batch_size = bottom[0]->shape()[0];

	//subtract (mu - t)
	kernel_label_subtract<Dtype> << <CAFFE_GET_BLOCKS(diff_.count()), CAFFE_CUDA_NUM_THREADS >> >(diff_.count(),
		data_dim + 2, class_size, data_dim, bottom_data, label, diff_.mutable_gpu_data());

	//square ( mu - t )^2
	caffe_gpu_mul(diff_.count(), diff_.gpu_data(), diff_.gpu_data(), diff_square_.mutable_gpu_data());

	//norm  : || mu-t || ^ 2
	kernel_diff_norm<Dtype> << <CAFFE_GET_BLOCKS(class_size * batch_size), CAFFE_CUDA_NUM_THREADS >> >(class_size * batch_size,
		class_size, data_dim, diff_square_.gpu_data(), diff_norm_.mutable_gpu_data());

	//calculate gaussian distribution
	kernel_normal_distribution<Dtype> << <CAFFE_GET_BLOCKS(class_size * batch_size), CAFFE_CUDA_NUM_THREADS >> >(class_size * batch_size,
		data_dim + 2, class_size, data_dim,
		diff_norm_.gpu_data(), bottom_data, alpha_pi_.mutable_gpu_data());

	Dtype norm_box[10];
	Dtype diff_box[90], diff_squre_box[90];
	Dtype bot_box[110], label_box[9];
	Dtype dist_box[10];
	Dtype sub;
	Dtype norm;
	for (int i = 0; i < batch_size; i++){
		hipMemcpy(diff_box, &diff_.gpu_data()[i * 90], sizeof(Dtype) * 90, hipMemcpyDeviceToHost);
		hipMemcpy(label_box, &label[i * 9], sizeof(Dtype) * 9, hipMemcpyDeviceToHost);
		hipMemcpy(bot_box, &bottom_data[110 * i], sizeof(Dtype) * 110, hipMemcpyDeviceToHost);
		hipMemcpy(diff_squre_box, &diff_square_.gpu_data()[i * 90], sizeof(Dtype) * 90, hipMemcpyDeviceToHost);
		hipMemcpy(norm_box, &diff_norm_.gpu_data()[i * 10], sizeof(Dtype) * 10, hipMemcpyDeviceToHost);
		hipMemcpy(dist_box, &alpha_pi_.gpu_data()[i * 10], sizeof(Dtype) * 10, hipMemcpyDeviceToHost);
		for (int j = 0; j < 110; j++)
			if (std::isnan(bot_box[j]) || std::isinf(bot_box[j]))
				printf("bottom data overflow.\n");
		for (int j = 0; j < 90; j++){
			int tClass_idx = j / 9;
			int internal_idx = j % 9;
			sub = bot_box[tClass_idx * 11 + internal_idx + 1] - label_box[j % 9];

			if (diff_box[j] != sub){
				printf("diff miss\n");
			}

			if (diff_squre_box[j] != (sub*sub))
				printf("square miss\n");

			if (std::isnan(diff_box[j]) || std::isinf(diff_box[j]))
				printf("diff_box data overflow.\n");
		}

		for (int j = 0; j < 10; j++){
			norm = 0;
			for (int k = 0; k < 9; k++)
				norm += diff_squre_box[j * 9 + k];
			if (norm != norm_box[j])
				printf("norm error\n");
		}
		for (int j = 0; j < 10; j++){
			Dtype alpha = bot_box[11 * j];
			Dtype sigma = bot_box[11 * j + 10];
			Dtype exp_gaussian = exp(-norm_box[j] / sigma / sigma / 2);
			Dtype dist_temp = alpha * exp_gaussian / pow(sigma, 9) / pow(2 * MATH_PI, 9 / 2);
			if (std::isnan(dist_box[j]) || std::isinf(dist_box[j]) || dist_box[j] < 0)
				printf("norm_box data overflow.\n");
			if (dist_temp != dist_box[j])
				printf("dist error\n");
		}
	}

	//sumation : ��(alpha * distribution)
	kernel_class_summation<Dtype> << <CAFFE_GET_BLOCKS(batch_size), CAFFE_CUDA_NUM_THREADS >> >(batch_size, class_size, alpha_pi_.gpu_data(), alpha_pi_sum_.mutable_gpu_data());

	Dtype alpha_pi_sum__box;
	for (int i = 0; i < batch_size; i++){
		hipMemcpy(&alpha_pi_sum__box, &alpha_pi_sum_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);
		hipMemcpy(dist_box, &alpha_pi_.gpu_data()[i * 10], sizeof(Dtype) * 10, hipMemcpyDeviceToHost);
		if (std::isnan(alpha_pi_sum__box) || std::isinf(alpha_pi_sum__box) || alpha_pi_sum__box < 0 || alpha_pi_sum__box == 0)
			printf("norm_box data overflow.\n");
	}

	//loss : ln ( sumation ) / number of batchsize
	Dtype loss;
	caffe_gpu_log(alpha_pi_sum_.count(), alpha_pi_sum_.gpu_data(), batch_loss_.mutable_gpu_data());
	caffe_gpu_dot(batch_loss_.count(), batch_loss_.gpu_data(), sum_multiplier_.gpu_data(), &loss);
	loss /= bottom[0]->num();
	top[0]->mutable_cpu_data()[0] = -loss;

	Dtype batch_loss_box;
	for (int i = 0; i < batch_size; i++){
		hipMemcpy(&batch_loss_box, &batch_loss_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);
		hipMemcpy(&alpha_pi_sum__box, &alpha_pi_sum_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);
		if (std::isnan(batch_loss_box) || std::isinf(batch_loss_box))
			printf("norm_box data overflow.\n");
	}

	if (std::isnan(loss) || std::isinf(loss)){
		printf("loss invalid value.\n");
	}
}

//Diff 0������ ���ְ� 1������ ����
template <typename Dtype>
void MDNLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	for (int i = 0; i < 2; ++i) {
		if (propagate_down[i]) {
			// i == 0 : bottom network i == 1 : label
			//��ȣ +- �ٽ� �ѹ� �����غ���
			Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
			const Dtype* bottom_data = bottom[i]->gpu_data();
			const int batch_size = bottom[0]->shape()[0];
			
			//calculate posterior probability ( alpha*pi / sumation ( alpha_i * pi_i )
			kernel_posterior_calc<Dtype> << <CAFFE_GET_BLOCKS(batch_size*class_size), CAFFE_CUDA_NUM_THREADS >> >
				(batch_size*class_size, batch_size, class_size, 
				alpha_pi_.gpu_data(), alpha_pi_sum_.gpu_data(), posterior_pi_.mutable_gpu_data());

			Dtype post_box[10], alpha_pi_box[10], alpha_pi_sum_box;
			Dtype sum_temp = 0, alpha_pi_sum_temp = 0;
			for (int i = 0; i < batch_size; i++){
				sum_temp = 0, alpha_pi_sum_temp = 0;
				hipMemcpy(post_box, &posterior_pi_.gpu_data()[i * 10], sizeof(Dtype) * 10, hipMemcpyDeviceToHost);
				hipMemcpy(alpha_pi_box, &alpha_pi_.gpu_data()[i * 10], sizeof(Dtype) * 10, hipMemcpyDeviceToHost);
				hipMemcpy(&alpha_pi_sum_box, &alpha_pi_sum_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);
				for (int j = 0; j < 10; j++){
					sum_temp += post_box[j];
					alpha_pi_sum_temp += alpha_pi_box[j];
				}
			}

			//calculate bottom diff (alpha_diff, mu_diff, sigma_diff)
			kernel_delta_calc<Dtype> << <CAFFE_GET_BLOCKS(bottom[i]->count()), CAFFE_CUDA_NUM_THREADS >> >(bottom[i]->count(),
				batch_size, class_size, data_dim + 2, data_dim, 
				posterior_pi_.gpu_data(), diff_.gpu_data(), diff_norm_.gpu_data(), bottom_data, bottom_diff);

			Dtype diff_box[110];
			for (int i = 0; i < batch_size; i++){
				hipMemcpy(diff_box, &bottom_diff[i*110], sizeof(Dtype) * 110, hipMemcpyDeviceToHost);
				for (int j = 0; j < 110; j++)
					if (std::isnan(diff_box[j]) || std::isinf(diff_box[j]))
						printf("loss invalid value.\n");
			}
		}
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(MDNLossLayer);

}  // namespace caffe
